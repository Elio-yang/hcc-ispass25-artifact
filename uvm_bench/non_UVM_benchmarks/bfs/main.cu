#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <cstring>
#include <iostream>
#include "graph.h"
#include "bfsCPU.h"
#include "bfsCUDA.cuh"

#include "../../common/file_op.h"

void runCpu(int startVertex, Graph &G, std::vector<int> &distance,
            std::vector<int> &parent, std::vector<bool> &visited) {
    printf("Starting sequential bfs.\n");
    auto start = std::chrono::steady_clock::now();
    bfsCPU(startVertex, G, distance, parent, visited);
    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time in milliseconds : %li ms.\n\n", duration);
}


#define checkError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }

}

int *d_adjacencyList;
int *d_edgesOffset;
int *d_edgesSize;
int *d_distance;
int *d_parent;
int *d_currentQueue;
int *d_nextQueue;
int *d_degrees;

int *incrDegrees;

// shared for all
float bt_malloc;
float bt_memcpy_h2d;
float bt_free;
float bt_memset;

// per kernel
float bt_malloc_simple;
float bt_memcpy_h2d_simple;
float bt_lauch_simple;
float bt_kernel_simple;
float bt_memcpy_d2h_simple;

float bt_malloc_queue;
float bt_memcpy_h2d_queue;
float bt_lauch_queue;
float bt_kernel_queue;
float bt_memcpy_d2h_queue;

float bt_malloc_scan;
float bt_memcpy_h2d_scan;
float bt_lauch_scan;
float bt_kernel_scan;
float bt_memcpy_d2h_scan;




void initCuda(Graph &G) {

    // checkError(hipMalloc((void **)&d_adjacencyList, G.numEdges * sizeof(int) ));
    // checkError(hipMalloc((void **)&d_edgesOffset, G.numVertices * sizeof(int) ));
    // checkError(hipMalloc((void **)&d_edgesSize, G.numVertices * sizeof(int)) );
    // checkError(hipMalloc((void **)&d_distance, G.numVertices * sizeof(int) ));
    // checkError(hipMalloc((void **)&d_parent, G.numVertices * sizeof(int) ));
    // checkError(hipMalloc((void **)&d_currentQueue, G.numVertices * sizeof(int) ));
    // checkError(hipMalloc((void **)&d_nextQueue, G.numVertices * sizeof(int) ));
    // checkError(hipMalloc((void **)&d_degrees, G.numVertices * sizeof(int) ));
    // checkError(hipHostMalloc((void **) &incrDegrees, sizeof(int) * G.numVertices));
    // checkError(hipMemcpy(d_adjacencyList, G.adjacencyList.data(), G.numEdges * sizeof(int), hipMemcpyHostToDevice));
    // checkError(hipMemcpy(d_edgesOffset, G.edgesOffset.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice));
    // checkError(hipMemcpy(d_edgesSize, G.edgesSize.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice ));
    clock_t t1 = clock();
    hipMalloc((void **)&d_adjacencyList, G.numEdges * sizeof(int) );
    hipMalloc((void **)&d_edgesOffset, G.numVertices * sizeof(int) );
    hipMalloc((void **)&d_edgesSize, G.numVertices * sizeof(int)) ;
    hipMalloc((void **)&d_distance, G.numVertices * sizeof(int) );
    hipMalloc((void **)&d_parent, G.numVertices * sizeof(int) );
    hipMalloc((void **)&d_currentQueue, G.numVertices * sizeof(int) );
    hipMalloc((void **)&d_nextQueue, G.numVertices * sizeof(int) );
    hipMalloc((void **)&d_degrees, G.numVertices * sizeof(int) );
    clock_t t2 = clock();
    hipHostMalloc((void **) &incrDegrees, sizeof(int) * G.numVertices);

    clock_t t3 = clock();
    hipMemcpy(d_adjacencyList, G.adjacencyList.data(), G.numEdges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgesOffset, G.edgesOffset.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgesSize, G.edgesSize.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice );
    clock_t t4 = clock();

    // shared for all
    bt_malloc = (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_memcpy_h2d = (float)(t4 - t3) / CLOCKS_PER_SEC;

    // per kernel
    bt_malloc_simple += bt_malloc;
    bt_memcpy_h2d_simple += bt_memcpy_h2d;

    bt_malloc_queue += bt_malloc;
    bt_memcpy_h2d_queue += bt_memcpy_h2d;
    
    bt_malloc_scan += bt_malloc;
    bt_memcpy_h2d_scan += bt_memcpy_h2d;


}

void finalizeCuda() {
    clock_t t1 = clock();
    hipFree(d_adjacencyList);
    hipFree(d_edgesOffset);
    hipFree(d_edgesSize);
    hipFree(d_distance);
    hipFree(d_parent);
    hipFree(d_currentQueue);
    hipFree(d_nextQueue);
    hipFree(d_degrees);
    clock_t t2 = clock();
    hipHostFree(incrDegrees);
    bt_free = (float)(t2 - t1) / CLOCKS_PER_SEC;
}



void checkOutput(std::vector<int> &distance, std::vector<int> &expectedDistance, Graph &G) {
    for (int i = 0; i < G.numVertices; i++) {
        if (distance[i] != expectedDistance[i]) {
            printf("%d %d %d\n", i, distance[i], expectedDistance[i]);
            printf("Wrong output!\n");
            exit(1);
        }
    }

    printf("Output OK!\n\n");
}


void initializeCudaBfs(int startVertex, std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //initialize values
    std::fill(distance.begin(), distance.end(), std::numeric_limits<int>::max());
    std::fill(parent.begin(), parent.end(), std::numeric_limits<int>::max());
    distance[startVertex] = 0;
    parent[startVertex] = 0;

    clock_t t1 = clock();
    hipMemcpy(d_distance, distance.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_parent, parent.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice);
    int firstElementQueue = startVertex;
    hipMemcpy(d_currentQueue, &firstElementQueue, sizeof(int), hipMemcpyHostToDevice);
    clock_t t2 = clock();
    // bt_memcpy_h2d = (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_memcpy_h2d_simple += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_memcpy_h2d_queue += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_memcpy_h2d_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
}

void finalizeCudaBfs(std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //copy memory from device
    clock_t t1 = clock();
    hipMemcpy(distance.data(), d_distance, G.numVertices * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(parent.data(), d_parent, G.numVertices * sizeof(int), hipMemcpyDeviceToHost);
    clock_t t2 = clock();
    // bt_memcpy_d2h = (float)(t2 - t1) / CLOCKS_PER_SEC;

    bt_memcpy_d2h_simple += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_memcpy_d2h_queue += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_memcpy_d2h_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
}

// gpu
void runCudaSimpleBfs(int startVertex, Graph &G, std::vector<int> &distance,
                      std::vector<int> &parent) {
    
    initializeCudaBfs(startVertex, distance, parent, G);


    int *changed;
    checkError(hipHostMalloc((void **) &changed, sizeof(int)));

    //launch kernel
    printf("Starting simple parallel bfs.\n");
    
    clock_t start_time = clock();
    *changed = 1;
    int level = 0;
    while (*changed) {
        *changed = 0;
        clock_t t1 = clock();
        simpleBfs<<<G.numVertices / 1024 + 1, 1024>>>(G.numVertices, level, d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance, d_parent, changed);                 
        clock_t t2 = clock();
        hipDeviceSynchronize();
        clock_t t3 = clock();
        level++;
        
        bt_lauch_simple += (float)(t2 - t1) / CLOCKS_PER_SEC;
        bt_kernel_simple += (float)(t3 - t2) / CLOCKS_PER_SEC;
    }
    clock_t end_time = clock();
    float duration = (float)(end_time - start_time) / CLOCKS_PER_SEC;
    
    printf("Elapsed time in : %f s.\n", duration);
    printf("My time in : %f s.\n", bt_kernel_simple + bt_lauch_simple);

    // FILE * fp = fopen("/shared/uvm_bench/log/simple-bfs.txt", "a");
	// if (fp == NULL) {
	// 	fprintf(stderr, "Error opening file!\n");
	// 	exit(1);
	// }
	// fprintf(fp, "%f\n", duration);
    // fclose(fp);
    finalizeCudaBfs(distance, parent, G);
}

// gpu
void runCudaQueueBfs(int startVertex, Graph &G, std::vector<int> &distance,
    std::vector<int> &parent) {
    initializeCudaBfs(startVertex, distance, parent, G);

    int *nextQueueSize;
    checkError(hipHostMalloc((void **)&nextQueueSize, sizeof(int)));
    //launch kernel
    printf("Starting queue parallel bfs.\n");

    clock_t start_time = clock();
    int queueSize = 1;
    *nextQueueSize = 0;
    int level = 0;
    while (queueSize) {
        clock_t t1 = clock();
        queueBfs<<<queueSize / 1024 + 1, 1024>>>(level, d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance, d_parent, queueSize,
                                                nextQueueSize, d_currentQueue, d_nextQueue);
        clock_t t2 = clock();
        hipDeviceSynchronize();
        clock_t t3 = clock();
        level++;
        queueSize = *nextQueueSize;
        *nextQueueSize = 0;
        std::swap(d_currentQueue, d_nextQueue);

        bt_lauch_queue += (float)(t2 - t1) / CLOCKS_PER_SEC;
        bt_kernel_queue += (float)(t3 - t2) / CLOCKS_PER_SEC;
    }
    clock_t end_time = clock();
    float duration = (float)(end_time - start_time) / CLOCKS_PER_SEC;

    printf("Elapsed time : %f s.\n", duration);
    printf("My time in : %f s.\n", bt_kernel_queue + bt_lauch_queue);

    // FILE * fp = fopen("/shared/uvm_bench/log/queue-bfs.txt", "a");
	// if (fp == NULL) {
	// 	fprintf(stderr, "Error opening file!\n");
	// 	exit(1);
	// }
	// fprintf(fp, "%f\n", duration);
    // fclose(fp);
    finalizeCudaBfs(distance, parent, G);
}

void nextLayer(int level, int queueSize) {
    clock_t t1 = clock();
    nextLayer<<<queueSize / 1024 + 1, 1024>>>(level, d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance, d_parent, queueSize,
                                            d_currentQueue);
    clock_t t2 = clock();
    hipDeviceSynchronize();
    clock_t t3 = clock();

    bt_lauch_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_kernel_scan += (float)(t3 - t2) / CLOCKS_PER_SEC;
}

void countDegrees(int level, int queueSize) {
    clock_t t1 = clock();
    countDegrees<<<queueSize / 1024 + 1, 1024>>>(d_adjacencyList, d_edgesOffset, d_edgesSize, d_parent, queueSize,
        d_currentQueue, d_degrees);
    clock_t t2 = clock();
    hipDeviceSynchronize();
    clock_t t3 = clock();

    bt_lauch_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_kernel_scan += (float)(t3 - t2) / CLOCKS_PER_SEC;

}

void scanDegrees(int queueSize) {
//run kernel so every block in d_currentQueue has prefix sums calculated
    clock_t t1 = clock();
    scanDegrees<<<queueSize / 1024 + 1, 1024>>>(queueSize, d_degrees, incrDegrees);
    clock_t t2 = clock();
    hipDeviceSynchronize();
    clock_t t3 = clock();

    bt_lauch_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_kernel_scan += (float)(t3 - t2) / CLOCKS_PER_SEC;

    //count prefix sums on CPU for ends of blocks exclusive
    //already written previous block sum
    incrDegrees[0] = 0;
    for (int i = 1024; i < queueSize + 1024; i += 1024) {
        incrDegrees[i / 1024] += incrDegrees[i / 1024 - 1];
    }
}

void assignVerticesNextQueue(int queueSize, int nextQueueSize) {
    clock_t t1 = clock();
    assignVerticesNextQueue<<<queueSize / 1024 + 1, 1024>>>(d_adjacencyList, d_edgesOffset, d_edgesSize, d_parent, queueSize, d_currentQueue,
        d_nextQueue, d_degrees, incrDegrees, nextQueueSize);
    clock_t t2 = clock();
    hipDeviceSynchronize();
    clock_t t3 = clock();

    bt_lauch_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_kernel_scan += (float)(t3 - t2) / CLOCKS_PER_SEC;
}

// gpu
void runCudaScanBfs(int startVertex, Graph &G, std::vector<int> &distance,
   std::vector<int> &parent) {
    initializeCudaBfs(startVertex, distance, parent, G);

    //launch kernel
    printf("Starting scan parallel bfs.\n");

    clock_t start_time = clock();
    int queueSize = 1;
    int nextQueueSize = 0;
    int level = 0;
    while (queueSize) {
        // next layer phase
        nextLayer(level, queueSize);
        // counting degrees phase
        countDegrees(level, queueSize);
        // doing scan on degrees
        scanDegrees(queueSize);
        nextQueueSize = incrDegrees[(queueSize - 1) / 1024 + 1];
        // assigning vertices to nextQueue
        assignVerticesNextQueue(queueSize, nextQueueSize);

        level++;
        queueSize = nextQueueSize;
        std::swap(d_currentQueue, d_nextQueue);
    }

    clock_t end_time = clock();
    float duration = (float)(end_time - start_time) / CLOCKS_PER_SEC;
   
    printf("Elapsed time: %f s.\n", duration);
    printf("My time in : %f s.\n", bt_kernel_scan + bt_lauch_scan);

    // FILE * fp = fopen("/shared/uvm_bench/log/scan-bfs.txt", "a");
	// if (fp == NULL) {
	// 	fprintf(stderr, "Error opening file!\n");
	// 	exit(1);
	// }
	// fprintf(fp, "%f\n", duration);
    // fclose(fp);


    finalizeCudaBfs(distance, parent, G);
}


int main(int argc, char **argv) {

    // read graph from standard input
    Graph G;
    int startVertex = atoi(argv[1]);
    readGraph(G, argc, argv);

    printf("Number of vertices %d\n", G.numVertices);
    printf("Number of edges %d\n\n", G.numEdges);

    //vectors for results
    std::vector<int> distance(G.numVertices, std::numeric_limits<int>::max());
    std::vector<int> parent(G.numVertices, std::numeric_limits<int>::max());
    std::vector<bool> visited(G.numVertices, false);

    //run CPU sequential bfs
    runCpu(startVertex, G, distance, parent, visited);

    //save results from sequential bfs
    std::vector<int> expectedDistance(distance);
    std::vector<int> expectedParent(parent);
   
    auto start = std::chrono::steady_clock::now();
    initCuda(G);
    //run CUDA simple parallel bfs
    runCudaSimpleBfs(startVertex, G, distance, parent);
    checkOutput(distance, expectedDistance, G);
    std::cout << distance[0] << std::endl;

    // // //run CUDA scan parallel bfs
    runCudaScanBfs(startVertex, G, distance, parent);
    //checkOutput(distance, expectedDistance, G);
    std::cout << distance[0] << std::endl;

    finalizeCuda();

	save_log("simple-bfs", "nor-brk", NULL, "%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f\n", bt_malloc_simple, bt_memcpy_h2d_simple, bt_lauch_simple, bt_kernel_simple, bt_memcpy_d2h_simple, bt_free, bt_memset);
	save_log("queue-bfs", "nor-brk", NULL, "%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f\n", bt_malloc_queue, bt_memcpy_h2d_queue, bt_lauch_queue, bt_kernel_queue, bt_memcpy_d2h_queue, bt_free, bt_memset);
	save_log("scan-bfs", "nor-brk", NULL, "%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f\n", bt_malloc_scan, bt_memcpy_h2d_scan, bt_lauch_scan, bt_kernel_scan, bt_memcpy_d2h_scan, bt_free, bt_memset);
    

    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Overall Elapsed time in milliseconds : %li ms.\n", duration);
    return 0;
}


