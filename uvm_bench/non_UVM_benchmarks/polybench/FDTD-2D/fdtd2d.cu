#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define GPU_DEVICE 0

/* Problem size */
#define tmax 500
#define NX 2048
#define NY 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_arrays(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
	int i, j;

  	for (i = 0; i < tmax; i++)
	{
		_fict_[i] = (DATA_TYPE) i;
	}
	
	for (i = 0; i < NX; i++)
	{
		for (j = 0; j < NY; j++)
		{
			ex[i*NY + j] = ((DATA_TYPE) i*(j+1) + 1) / NX;
			ey[i*NY + j] = ((DATA_TYPE) (i-1)*(j+2) + 2) / NX;
			hz[i*NY + j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
		}
	}
}


void runFdtd(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
	int t, i, j;
	
	for (t=0; t < tmax; t++)  
	{
		for (j=0; j < NY; j++)
		{
			ey[0*NY + j] = _fict_[t];
		}
	
		for (i = 1; i < NX; i++)
		{
       		for (j = 0; j < NY; j++)
			{
       			ey[i*NY + j] = ey[i*NY + j] - 0.5*(hz[i*NY + j] - hz[(i-1)*NY + j]);
        		}
		}

		for (i = 0; i < NX; i++)
		{
       		for (j = 1; j < NY; j++)
			{
				ex[i*(NY+1) + j] = ex[i*(NY+1) + j] - 0.5*(hz[i*NY + j] - hz[i*NY + (j-1)]);
			}
		}

		for (i = 0; i < NX; i++)
		{
			for (j = 0; j < NY; j++)
			{
				hz[i*NY + j] = hz[i*NY + j] - 0.7*(ex[i*(NY+1) + (j+1)] - ex[i*(NY+1) + j] + ey[(i+1)*NY + j] - ey[i*NY + j]);
			}
		}
	}
}


void compareResults(DATA_TYPE* hz1, DATA_TYPE* hz2)
{
	int i, j, fail;
	fail = 0;
	
	for (i=0; i < NX; i++) 
	{
		for (j=0; j < NY; j++) 
		{
			if (percentDiff(hz1[i*NY + j], hz2[i*NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}



__global__ void fdtd_step1_kernel(DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY))
	{
		if (i == 0) 
		{
			ey[i * NY + j] = _fict_[t];
		}
		else
		{ 
			ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
		}
	}
}



__global__ void fdtd_step2_kernel(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY) && (j > 0))
	{
		ex[i * (NY+1) + j] = ex[i * (NY+1) + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
	}
}


__global__ void fdtd_step3_kernel(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY))
	{	
		hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * (NY+1) + (j+1)] - ex[i * (NY+1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
	}
}


void fdtdCuda(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz, DATA_TYPE* hz_outputFromGpu)
{
	double t_start, t_end;

	DATA_TYPE *_fict_gpu;
	DATA_TYPE *ex_gpu;
	DATA_TYPE *ey_gpu;
	DATA_TYPE *hz_gpu;

	double t_malloc_s, t_malloc_e, t_memcpy_h2d, t_memcpy_d2h, t_kernel, t_lauch, t_free, t_memset;

	t_memset = 0.0;

	double s1 = rtclock();
	hipMalloc((void **)&_fict_gpu, sizeof(DATA_TYPE) * tmax);
	hipMalloc((void **)&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
	hipMalloc((void **)&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
	hipMalloc((void **)&hz_gpu, sizeof(DATA_TYPE) * NX * NY);
	double s2 = rtclock();
	hipMemcpy(_fict_gpu, _fict_, sizeof(DATA_TYPE) * tmax, hipMemcpyHostToDevice);
	hipMemcpy(ex_gpu, ex, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemcpyHostToDevice);
	hipMemcpy(ey_gpu, ey, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemcpyHostToDevice);
	hipMemcpy(hz_gpu, hz, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
	double s3 = rtclock();


	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)), (size_t)ceil(((float)NX) / ((float)block.y)));

	double t4, t5, t6, t7, t8, t9;
	for(int t = 0; t< tmax; t++)
	{
		t_start = rtclock();
		fdtd_step1_kernel<<<grid,block>>>(_fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
		t4 = rtclock();
		hipDeviceSynchronize();
		t5 = rtclock();
		fdtd_step2_kernel<<<grid,block>>>(ex_gpu, ey_gpu, hz_gpu, t);
		t6 = rtclock();
		hipDeviceSynchronize();
		t7 = rtclock();
		fdtd_step3_kernel<<<grid,block>>>(ex_gpu, ey_gpu, hz_gpu, t);
		t8 = rtclock();
		hipDeviceSynchronize();
		t9 = rtclock();

		t_lauch += ((t4-t_start) + (t6-t5) + (t8-t7));
		t_kernel += ((t5-t4) + (t7-t6) + (t9-t8));
	}
    // fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	// save_log(__FILE__,"nor",NULL, "%0.6lf\n", t_end - t_start);
	double t = rtclock();
	hipMemcpy(hz_outputFromGpu, hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemcpyDeviceToHost);	
	double t10 = rtclock();
	hipFree(_fict_gpu);
	hipFree(ex_gpu);
	hipFree(ey_gpu);
	hipFree(hz_gpu);
	double t11 = rtclock();

	// =================================================================
	double bt_malloc = s2-s1;
	double bt_memcpy_h2d = s3-s2;
	double bt_lauch = t_lauch;
	double bt_kernel = t_kernel;
	double bt_memcpy_d2h = t10 - t;
	double bt_free = t11 - t10;
	double bt_memset = 0.0;
	save_log(__FILE__, "nor-brk", NULL, "%0.6lf,%0.6lf,%0.6lf,%0.6lf,%0.6lf,%0.6lf,%0.6lf\n", bt_malloc, bt_memcpy_h2d, bt_lauch, bt_kernel, bt_memcpy_d2h, bt_free, bt_memset);

}


int main()
{
	double t_start, t_end;

	DATA_TYPE* _fict_;
	DATA_TYPE* ex;
	DATA_TYPE* ey;
	DATA_TYPE* hz;
	DATA_TYPE* hz_outputFromGpu;

	// _fict_ = (DATA_TYPE*)malloc(tmax*sizeof(DATA_TYPE));
	// ex = (DATA_TYPE*)malloc(NX*(NY+1)*sizeof(DATA_TYPE));
	// ey = (DATA_TYPE*)malloc((NX+1)*NY*sizeof(DATA_TYPE));
	// hz = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
	// hz_outputFromGpu = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));


	hipHostMalloc((void **)&_fict_,tmax*sizeof(DATA_TYPE));
	hipHostMalloc((void **)&ex,NX*(NY+1)*sizeof(DATA_TYPE));
	hipHostMalloc((void **)&ey,(NX+1)*NY*sizeof(DATA_TYPE));
	hipHostMalloc((void **)&hz,NX*NY*sizeof(DATA_TYPE));
	hipHostMalloc((void **)&hz_outputFromGpu,NX*NY*sizeof(DATA_TYPE));


	init_arrays(_fict_, ex, ey, hz);

	GPU_argv_init();
	fdtdCuda(_fict_, ex, ey, hz, hz_outputFromGpu);

	// t_start = rtclock();
	// runFdtd(_fict_, ex, ey, hz);
	// t_end = rtclock();
	
	// fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	// compareResults(hz, hz_outputFromGpu);

	printf("hz[0] = %f\n", hz_outputFromGpu[0]);

	hipFree(_fict_);
	hipFree(ex);
	hipFree(ey);
	hipFree(hz);
	hipFree(hz_outputFromGpu);

	return 0;
}

