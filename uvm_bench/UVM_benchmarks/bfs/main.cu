#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <cstring>
#include "graph.h"
#include "bfsCPU.h"
#include "bfsCUDA.cuh"
#include "../../common/file_op.h"


void runCpu(int startVertex, Graph &G, std::vector<int> &distance,
            std::vector<int> &parent, std::vector<bool> &visited) {
    printf("Starting sequential bfs.\n");
    auto start = std::chrono::steady_clock::now();
    bfsCPU(startVertex, G, distance, parent, visited);
    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time in milliseconds : %li ms.\n\n", duration);
}


#define checkError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }

}

int *u_adjacencyList;
int *u_edgesOffset;
int *u_edgesSize;
int *u_distance;
int *u_parent;
int *u_currentQueue;
int *u_nextQueue;
int *u_degrees;
int *incrDegrees;


// shared for all
float bt_malloc;
float bt_memcpy_h2d;
float bt_free;
float bt_memset;

// per kernel
// float bt_malloc_simple;
// float bt_lauch_simple;
// // float bt_lauch2_simple;
// float bt_kernel_simple;
// // float bt_kernel2_simple;
// float paging_simple;

// float bt_malloc_queue;
// float bt_lauch_queue;
// float bt_kernel_queue;
// float paging_queue;

// float bt_malloc_scan;
// float bt_lauch_scan;
// float bt_kernel_scan;
// float paging_scan;

float bt_malloc_simple;
float bt_lauch_simple;
float bt_kernel_simple;
//not used
float bt_memcpy_d2h_simple;
float bt_memcpy_h2d_simple;

float bt_malloc_queue;
float bt_memcpy_h2d_queue;
float bt_lauch_queue;
float bt_kernel_queue;
float bt_memcpy_d2h_queue;

float bt_malloc_scan;
float bt_memcpy_h2d_scan;
float bt_lauch_scan;
float bt_kernel_scan;
float bt_memcpy_d2h_scan;

void initCuda(Graph &G) {
    // checkError(hipMallocManaged(&u_adjacencyList, G.numEdges * sizeof(int) ));
    // checkError(hipMallocManaged(&u_edgesOffset, G.numVertices * sizeof(int) ));
    // checkError(hipMallocManaged(&u_edgesSize, G.numVertices * sizeof(int)) );
    // checkError(hipMallocManaged(&u_distance, G.numVertices * sizeof(int) ));
    // checkError(hipMallocManaged(&u_parent, G.numVertices * sizeof(int) ));
    // checkError(hipMallocManaged(&u_currentQueue, G.numVertices * sizeof(int) ));
    // checkError(hipMallocManaged(&u_nextQueue, G.numVertices * sizeof(int) ));
    // checkError(hipMallocManaged(&u_degrees, G.numVertices * sizeof(int) ));



    // checkError(hipHostMalloc((void **) &incrDegrees, sizeof(int) * G.numVertices));
    clock_t t1 = clock();
    hipMallocManaged(&u_adjacencyList, G.numEdges * sizeof(int));
    hipMallocManaged(&u_edgesOffset, G.numVertices * sizeof(int));
    hipMallocManaged(&u_edgesSize, G.numVertices * sizeof(int));
    hipMallocManaged(&u_distance, G.numVertices * sizeof(int));
    hipMallocManaged(&u_parent, G.numVertices * sizeof(int));
    hipMallocManaged(&u_currentQueue, G.numVertices * sizeof(int));
    hipMallocManaged(&u_nextQueue, G.numVertices * sizeof(int));
    hipMallocManaged(&u_degrees, G.numVertices * sizeof(int));
    clock_t t2 = clock();

    bt_malloc = (float)(t2 - t1) / CLOCKS_PER_SEC;
    
    bt_malloc_simple += bt_malloc;
    bt_malloc_queue += bt_malloc;
    bt_malloc_scan += bt_malloc;


    hipHostMalloc((void **)&incrDegrees, sizeof(int) * G.numVertices);
    memcpy(u_adjacencyList, G.adjacencyList.data(), G.numEdges * sizeof(int));
    memcpy(u_edgesOffset, G.edgesOffset.data(), G.numVertices * sizeof(int));
    memcpy(u_edgesSize, G.edgesSize.data(), G.numVertices * sizeof(int));
    // checkError(hipMemcpy(d_adjacencyList, G.adjacencyList.data(), G.numEdges * sizeof(int), hipMemcpyHostToDevice));
    // checkError(hipMemcpy(d_edgesOffset, G.edgesOffset.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice));
    // checkError(hipMemcpy(d_edgesSize, G.edgesSize.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice ));

}

void finalizeCuda() {
    clock_t t1 = clock();
    hipFree(u_adjacencyList);
    hipFree(u_edgesOffset);
    hipFree(u_edgesSize);
    hipFree(u_distance);
    hipFree(u_parent);
    hipFree(u_currentQueue);
    hipFree(u_nextQueue);
    hipFree(u_degrees);
    clock_t t2 = clock();
    bt_free = (float)(t2 - t1) / CLOCKS_PER_SEC;

    hipHostFree(incrDegrees);

}



void checkOutput(std::vector<int> &distance, std::vector<int> &expectedDistance, Graph &G) {
    for (int i = 0; i < G.numVertices; i++) {
        if (*(u_distance+i) != expectedDistance[i]) {
            printf("%d %d %d\n", i, distance[i], expectedDistance[i]);
            printf("Wrong output!\n");
            exit(1);
        }
    }

    printf("Output OK!\n\n");
}


void initializeCudaBfs(int startVertex, std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //initialize values
    std::fill(distance.begin(), distance.end(), std::numeric_limits<int>::max());
    std::fill(parent.begin(), parent.end(), std::numeric_limits<int>::max());
    distance[startVertex] = 0;
    parent[startVertex] = 0;

    // checkError(hipMemcpy(d_distance, distance.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice));
    // checkError(hipMemcpy(d_parent, parent.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice));
    memcpy(u_distance, distance.data(), G.numVertices * sizeof(int));
    memcpy(u_parent, parent.data(), G.numVertices * sizeof(int));

    int firstElementQueue = startVertex;
    // hipMemcpy(d_currentQueue, &firstElementQueue, sizeof(int), hipMemcpyHostToDevice);
    *u_currentQueue = firstElementQueue;
}




void finalizeCudaBfs(std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //copy memory from device
    // checkError(hipMemcpy(distance.data(), d_distance, G.numVertices * sizeof(int), hipMemcpyDeviceToHost));
    // checkError(hipMemcpy(parent.data(), d_parent, G.numVertices * sizeof(int), hipMemcpyDeviceToHost));
}

void runCudaSimpleBfs(int startVertex, Graph &G, std::vector<int> &distance,
                      std::vector<int> &parent) {
    initializeCudaBfs(startVertex, distance, parent, G);

    int *changed;
    checkError(hipHostMalloc((void **) &changed, sizeof(int)));

    //launch kernel
    printf("Starting simple parallel bfs.\n");
    // auto start = std::chrono::steady_clock::now();
    // ============================================================
    *changed = 1;
    int level = 0;
    while (*changed) {
        *changed = 0;
        clock_t t1 = clock();
        simpleBfs<<<G.numVertices / 1024 + 1, 1024>>>(G.numVertices, level, u_adjacencyList, u_edgesOffset, u_edgesSize, u_distance, u_parent, changed);                 
        clock_t t2 = clock();
        hipDeviceSynchronize();
        clock_t t3 = clock();
        bt_lauch_simple += (float)(t2 - t1) / CLOCKS_PER_SEC;
        bt_kernel_simple += (float)(t3 - t2) / CLOCKS_PER_SEC;
        level++;
    }    
    // // second run to getridof paging
    // *changed = 1;
    // level = 0;
    // while (*changed) {
    //     *changed = 0;
    //     clock_t t1 = clock();
    //     simpleBfs<<<G.numVertices / 1024 + 1, 1024>>>(G.numVertices, level, u_adjacencyList, u_edgesOffset, u_edgesSize, u_distance, u_parent, changed);                 
    //     clock_t t2 = clock();
    //     hipDeviceSynchronize();
    //     clock_t t3 = clock();
    //     bt_lauch2_simple += (float)(t2 - t1) / CLOCKS_PER_SEC;
    //     bt_kernel2_simple += (float)(t3 - t2) / CLOCKS_PER_SEC;
    //     level++;
    // }
    printf("Elapsed time 1: %f s.\n", bt_kernel_simple+bt_lauch_simple);
    // printf("Elapsed time 2: %f s.\n", bt_kernel2_simple+bt_lauch2_simple);
    // auto end = std::chrono::steady_clock::now();
    // long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time : %f s.\n", duration);
    // FILE * fp = fopen("/shared/uvm_bench/log/simple-bfs-uvm.txt", "a");
	// if (fp == NULL) {
	// 	fprintf(stderr, "Error opening file!\n");
	// 	exit(1);
	// }
	// fprintf(fp, "%f\n", duration);
    // fclose(fp);
    finalizeCudaBfs(distance, parent, G);
}


void runCudaQueueBfs(int startVertex, Graph &G, std::vector<int> &distance,
    std::vector<int> &parent) {
    initializeCudaBfs(startVertex, distance, parent, G);

    int *nextQueueSize;
    checkError(hipHostMalloc((void **)&nextQueueSize, sizeof(int)));
    //launch kernel
    printf("Starting queue parallel bfs.\n");
    // auto start = std::chrono::steady_clock::now();
    clock_t start = clock();
    int queueSize = 1;
    *nextQueueSize = 0;
    int level = 0;
    while (queueSize) {
        clock_t t1 = clock();
        queueBfs<<<queueSize / 1024 + 1, 1024>>>(level, u_adjacencyList, u_edgesOffset, u_edgesSize, u_distance, u_parent, queueSize,
                                                nextQueueSize, u_currentQueue, u_nextQueue);
        clock_t t2 = clock();
        hipDeviceSynchronize();
        clock_t t3 = clock();

        level++;
        queueSize = *nextQueueSize;
        *nextQueueSize = 0;
        std::swap(u_currentQueue, u_nextQueue);
        bt_lauch_queue += (float)(t2 - t1) / CLOCKS_PER_SEC;
        bt_kernel_queue += (float)(t3 - t2) / CLOCKS_PER_SEC;        
    }
    clock_t end = clock();
    float duration = (float)(end - start) / CLOCKS_PER_SEC;
    // auto end = std::chrono::steady_clock::now();
    // long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time: %f s.\n", duration);
    printf("My time in : %f s.\n", bt_kernel_queue + bt_lauch_queue);
    // FILE * fp = fopen("/shared/uvm_bench/log/queue-bfs-uvm.txt", "a");
	// if (fp == NULL) {
	// 	fprintf(stderr, "Error opening file!\n");
	// 	exit(1);
	// }
	// fprintf(fp, "%f\n", duration);
    // fclose(fp);

    finalizeCudaBfs(distance, parent, G);
}

void nextLayer(int level, int queueSize) {
    clock_t t1 = clock();
    nextLayer<<<queueSize / 1024 + 1, 1024>>>(level, u_adjacencyList, u_edgesOffset, u_edgesSize, u_distance, u_parent, queueSize,
                                            u_currentQueue);
    clock_t t2 = clock();
    hipDeviceSynchronize();
    clock_t t3 = clock();
    bt_lauch_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_kernel_scan += (float)(t3 - t2) / CLOCKS_PER_SEC;

}

void countDegrees(int level, int queueSize) {
    clock_t t1 = clock();
    countDegrees<<<queueSize / 1024 + 1, 1024>>>(u_adjacencyList, u_edgesOffset, u_edgesSize, u_parent, queueSize,
        u_currentQueue, u_degrees);
    clock_t t2 = clock();
    hipDeviceSynchronize();
    clock_t t3 = clock();

    bt_lauch_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_kernel_scan += (float)(t3 - t2) / CLOCKS_PER_SEC;    

}

void scanDegrees(int queueSize) {
//run kernel so every block in d_currentQueue has prefix sums calculated
    clock_t t1 = clock();
    scanDegrees<<<queueSize / 1024 + 1, 1024>>>(queueSize, u_degrees, incrDegrees);
    clock_t t2 = clock();
    hipDeviceSynchronize();
    clock_t t3 = clock();

    bt_lauch_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_kernel_scan += (float)(t3 - t2) / CLOCKS_PER_SEC;

    //count prefix sums on CPU for ends of blocks exclusive
    //already written previous block sum
    incrDegrees[0] = 0;
    for (int i = 1024; i < queueSize + 1024; i += 1024) {
        incrDegrees[i / 1024] += incrDegrees[i / 1024 - 1];
    }
}

void assignVerticesNextQueue(int queueSize, int nextQueueSize) {
    clock_t t1 = clock();
    assignVerticesNextQueue<<<queueSize / 1024 + 1, 1024>>>(u_adjacencyList, u_edgesOffset, u_edgesSize, u_parent, queueSize, u_currentQueue,
        u_nextQueue, u_degrees, incrDegrees, nextQueueSize);
    clock_t t2 = clock();
    hipDeviceSynchronize();
    clock_t t3 = clock();
    
    bt_lauch_scan += (float)(t2 - t1) / CLOCKS_PER_SEC;
    bt_kernel_scan += (float)(t3 - t2) / CLOCKS_PER_SEC;

}

void runCudaScanBfs(int startVertex, Graph &G, std::vector<int> &distance,
   std::vector<int> &parent) {
    initializeCudaBfs(startVertex, distance, parent, G);

    //launch kernel
    printf("Starting scan parallel bfs.\n");
    // auto start = std::chrono::steady_clock::now();

    clock_t start = clock();
    int queueSize = 1;
    int nextQueueSize = 0;
    int level = 0;
    while (queueSize) {
        // next layer phase
        nextLayer(level, queueSize);
        // counting degrees phase
        countDegrees(level, queueSize);
        // doing scan on degrees
        scanDegrees(queueSize);
        nextQueueSize = incrDegrees[(queueSize - 1) / 1024 + 1];
        // assigning vertices to nextQueue
        assignVerticesNextQueue(queueSize, nextQueueSize);

        level++;
        queueSize = nextQueueSize;
        std::swap(u_currentQueue, u_nextQueue);
    }
    clock_t end = clock();
    float duration = (float)(end - start) / CLOCKS_PER_SEC;

    // auto end = std::chrono::steady_clock::now();
    // long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time : %f s.\n", duration);
    printf("My time in : %f s.\n", bt_kernel_scan + bt_lauch_scan);

    // FILE * fp = fopen("/shared/uvm_bench/log/scan-bfs-uvm.txt", "a");
	// if (fp == NULL) {
	// 	fprintf(stderr, "Error opening file!\n");
	// 	exit(1);
	// }
	// fprintf(fp, "%f\n", duration);
    // fclose(fp);

    finalizeCudaBfs(distance, parent, G);
}


int main(int argc, char **argv) {

    // read graph from standard input
    Graph G;
    int startVertex = atoi(argv[1]);
    readGraph(G, argc, argv);

    printf("Number of vertices %d\n", G.numVertices);
    printf("Number of edges %d\n\n", G.numEdges);

    //vectors for results
    std::vector<int> distance(G.numVertices, std::numeric_limits<int>::max());
    std::vector<int> parent(G.numVertices, std::numeric_limits<int>::max());
    std::vector<bool> visited(G.numVertices, false);

    //run CPU sequential bfs
    runCpu(startVertex, G, distance, parent, visited);

    //save results from sequential bfs
    std::vector<int> expectedDistance(distance);
    std::vector<int> expectedParent(parent);
    auto start = std::chrono::steady_clock::now();
    initCuda(G);
    //run CUDA simple parallel bfs
    runCudaSimpleBfs(startVertex, G, distance, parent);
    checkOutput(distance, expectedDistance, G);

    // //run CUDA queue parallel bfs
    runCudaQueueBfs(startVertex, G, distance, parent);
    checkOutput(distance, expectedDistance, G);

    // //run CUDA scan parallel bfs
    runCudaScanBfs(startVertex, G, distance, parent);
    checkOutput(distance, expectedDistance, G);
    finalizeCuda();


    save_log("simple-bfs", "uvm-brk", NULL, "%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f\n", bt_malloc_simple, bt_memcpy_h2d_simple, bt_lauch_simple, bt_kernel_simple, bt_memcpy_d2h_simple, bt_free, bt_memset);
	save_log("queue-bfs", "uvm-brk", NULL, "%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f\n", bt_malloc_queue, bt_memcpy_h2d_queue, bt_lauch_queue, bt_kernel_queue, bt_memcpy_d2h_queue, bt_free, bt_memset);
	save_log("scan-bfs", "uvm-brk", NULL, "%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f,%0.6f\n", bt_malloc_scan, bt_memcpy_h2d_scan, bt_lauch_scan, bt_kernel_scan, bt_memcpy_d2h_scan, bt_free, bt_memset);
    

    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Overall Elapsed time in milliseconds : %li ms.\n", duration);
    return 0;
}


