#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define GPU_DEVICE 0

/* Problem size */
#define tmax 500
#define NX 2048
#define NY 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

double t_lauch;
double t_kernel;


void init_arrays(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz, DATA_TYPE* _fict_gpu, DATA_TYPE* ex_gpu, DATA_TYPE* ey_gpu, DATA_TYPE* hz_gpu)
{
	int i, j;

  	for (i = 0; i < tmax; i++)
	{
		_fict_[i] = (DATA_TYPE) i;
		_fict_gpu[i] = (DATA_TYPE) i;
	}
	
	for (i = 0; i < NX; i++)
	{
		for (j = 0; j < NY; j++)
		{
			ex[i*NY + j] = ((DATA_TYPE) i*(j+1) + 1) / NX;
			ey[i*NY + j] = ((DATA_TYPE) (i-1)*(j+2) + 2) / NX;
			hz[i*NY + j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
			ex_gpu[i*NY + j] = ((DATA_TYPE) i*(j+1) + 1) / NX;
			ey_gpu[i*NY + j] = ((DATA_TYPE) (i-1)*(j+2) + 2) / NX;
			hz_gpu[i*NY + j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
		}
	}
}


void runFdtd(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
	int t, i, j;
	
	for (t=0; t < tmax; t++)  
	{
		for (j=0; j < NY; j++)
		{
			ey[0*NY + j] = _fict_[t];
		}
	
		for (i = 1; i < NX; i++)
		{
       		for (j = 0; j < NY; j++)
			{
       			ey[i*NY + j] = ey[i*NY + j] - 0.5*(hz[i*NY + j] - hz[(i-1)*NY + j]);
        		}
		}

		for (i = 0; i < NX; i++)
		{
       		for (j = 1; j < NY; j++)
			{
				ex[i*(NY+1) + j] = ex[i*(NY+1) + j] - 0.5*(hz[i*NY + j] - hz[i*NY + (j-1)]);
			}
		}

		for (i = 0; i < NX; i++)
		{
			for (j = 0; j < NY; j++)
			{
				hz[i*NY + j] = hz[i*NY + j] - 0.7*(ex[i*(NY+1) + (j+1)] - ex[i*(NY+1) + j] + ey[(i+1)*NY + j] - ey[i*NY + j]);
			}
		}
	}
}


void compareResults(DATA_TYPE* hz1, DATA_TYPE* hz2)
{
	int i, j, fail;
	fail = 0;
	
	for (i=0; i < NX; i++) 
	{
		for (j=0; j < NY; j++) 
		{
			if (percentDiff(hz1[i*NY + j], hz2[i*NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

__global__ void fdtd_step1_kernel(DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY))
	{
		if (i == 0) 
		{
			ey[i * NY + j] = _fict_[t];
		}
		else
		{ 
			ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
		}
	}
}

__global__ void fdtd_step2_kernel(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY) && (j > 0))
	{
		ex[i * (NY+1) + j] = ex[i * (NY+1) + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
	}
}


__global__ void fdtd_step3_kernel(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY))
	{	
		hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * (NY+1) + (j+1)] - ex[i * (NY+1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
	}
}


void fdtdCuda(DATA_TYPE* _fict_gpu, DATA_TYPE* ex_gpu, DATA_TYPE* ey_gpu, DATA_TYPE* hz_gpu)
{
	double t_start, t_end;
	double t4, t5, t6, t7, t8, t9;


	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)), (size_t)ceil(((float)NX) / ((float)block.y)));


	for(int t = 0; t< tmax; t++)
	{	
		t_start = rtclock();
		fdtd_step1_kernel<<<grid,block>>>(_fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
		t4 = rtclock();
		hipDeviceSynchronize();
		t5 = rtclock();
		fdtd_step2_kernel<<<grid,block>>>(ex_gpu, ey_gpu, hz_gpu, t);
		t6 = rtclock();
		hipDeviceSynchronize();
		t7 = rtclock();
		fdtd_step3_kernel<<<grid,block>>>(ex_gpu, ey_gpu, hz_gpu, t);
		t8 = rtclock();
		hipDeviceSynchronize();
		t9 = rtclock();

		t_lauch += ((t4-t_start) + (t6-t5) + (t8-t7));
		t_kernel += ((t5-t4) + (t7-t6) + (t9-t8));
	}
	
	// t_end = rtclock();
    fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_lauch + t_kernel);
}


int main()
{
	double t_start, t_end;

	DATA_TYPE* _fict_;
	DATA_TYPE* ex;
	DATA_TYPE* ey;
	DATA_TYPE* hz;

	DATA_TYPE *_fict_gpu;
	DATA_TYPE *ex_gpu;
	DATA_TYPE *ey_gpu;
	DATA_TYPE *hz_gpu;

	_fict_ = (DATA_TYPE*)malloc(tmax*sizeof(DATA_TYPE));
	ex = (DATA_TYPE*)malloc(NX*(NY+1)*sizeof(DATA_TYPE));
	ey = (DATA_TYPE*)malloc((NX+1)*NY*sizeof(DATA_TYPE));
	hz = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));

	double s1 = rtclock();
	hipMallocManaged(&_fict_gpu, sizeof(DATA_TYPE) * tmax);
	hipMallocManaged(&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
	hipMallocManaged(&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
	hipMallocManaged(&hz_gpu, sizeof(DATA_TYPE) * NX * NY);
	double s2 = rtclock();

	init_arrays(_fict_, ex, ey, hz, _fict_gpu, ex_gpu, ey_gpu, hz_gpu);

	GPU_argv_init();
	
	fdtdCuda(_fict_gpu, ex_gpu, ey_gpu, hz_gpu);

	// t_start = rtclock();
	// runFdtd(_fict_, ex, ey, hz);
	// t_end = rtclock();
	
	// fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	// compareResults(hz, hz_gpu);

	printf("hz[0]: %f\n", hz_gpu[0]);

	free(_fict_);
	free(ex);
	free(ey);
	free(hz);
	
	double t10 = rtclock();
	hipFree(_fict_gpu);
	hipFree(ex_gpu);
	hipFree(ey_gpu);
	hipFree(hz_gpu);
	double t11 = rtclock();

	// =================================================================
	double bt_malloc = s2-s1;
	double bt_memcpy_h2d = 0.0;
	double bt_lauch = t_lauch;
	double bt_kernel = t_kernel;
	double bt_memcpy_d2h = 0.0;
	double bt_free = t11 - t10;
	double bt_memset = 0.0;
	save_log(__FILE__, "uvm-brk", NULL, "%0.6lf,%0.6lf,%0.6lf,%0.6lf,%0.6lf,%0.6lf,%0.6lf\n", bt_malloc, bt_memcpy_h2d, bt_lauch, bt_kernel, bt_memcpy_d2h, bt_free, bt_memset);

	return 0;
}

